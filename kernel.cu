/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 256
#define TILE_WIDTH 16

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE
    __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;  int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int Row = by * blockDim.y + ty;
    int Col = bx * blockDim.x + tx;
    float Cvalue = 0;

    for (int p = 0; p < (k-1) / TILE_WIDTH + 1; ++p) {  
        // loading of M and N tiles into shared memory
        if(Row < m && p * TILE_WIDTH+tx < k) {  // Boundary condition for M tile
            ds_A[ty][tx] = A[Row * k + p * TILE_WIDTH + tx];
        } else {
            ds_A[ty][tx] = 0.0;
        }
        if (p*TILE_WIDTH+ty < k && Col < n) {  // Boundary condition for N tile
            ds_B[ty][tx] = B[(p * TILE_WIDTH + ty) * n + Col];
        } else {
            ds_B[ty][tx] = 0.0;
        }
        __syncthreads();
        
        for (int i = 0; i < TILE_WIDTH; ++i) {
            Cvalue += ds_A[ty][i] * ds_B[i][tx];
           
            __syncthreads();
        }
    }
    if (Row < m && Col < n) {
         C[Row * n + Col] = Cvalue;
    }
}



void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n')) {
	printf("unsupported value of 'transa'\n");
    	return;
    }

    if ((transb != 'N') && (transb != 'n')) {
	printf("unsupported value of 'transb'\n");
	return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
	printf("unsupported value of alpha\n");
	return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
	printf("unsupported value of beta\n");
	return;
    }

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;

    //INSERT CODE HERE
    dim3 DimGrid((n-1)/TILE_WIDTH + 1, (m-1)/TILE_WIDTH + 1, 1);
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);



    // Invoke CUDA kernel -----------------------------------------------------

    //INSERT CODE HERE
    mysgemm<<<DimGrid,DimBlock>>>(m, n, k, A, B, C);  



}


